#include "hip/hip_runtime.h"
#include "kernel.cuh"

__constant__ size_t CUDA_minChunkY;
__constant__ size_t CUDA_lookupSize;

__device__ Vec3* CUDA_vertices;
__device__ Vec2* CUDA_texCoords;
__device__ Triangle* CUDA_triangles;

__device__ color* CUDA_lookupColors;
__device__ uint16_t* CUDA_lookupIndices;

namespace CUDA {

	void setMinChunkY(size_t chunkY) {
		checkCUDA(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_minChunkY), (const void*)&chunkY, sizeof(size_t), 0, hipMemcpyHostToDevice));
	}

	void setLookupSize(size_t size) {
		checkCUDA(hipMemcpyToSymbol(HIP_SYMBOL(CUDA_lookupSize), (const void*)&size, sizeof(size_t), 0, hipMemcpyHostToDevice));
	}

	hipTextureObject_t* createTexture(Image& img) {

		hipChannelFormatDesc channelDesc = hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned);
		hipArray_t cuArray;
		hipMallocArray(&cuArray, &channelDesc, img.width, img.height);

		const size_t spitch = (size_t)img.width * img.numChannels * sizeof(uint8_t);

		size_t texAtlasSize = 0;
		const uint8_t* pixels = img.pixels(texAtlasSize);
		hipMemcpy2DToArray(cuArray, 0, 0, pixels, spitch, spitch, img.height, hipMemcpyHostToDevice);

		hipResourceDesc resDesc{ hipResourceTypeArray, { cuArray } };

		hipTextureDesc texDesc;
		memset(&texDesc, 0, sizeof(texDesc));
		texDesc.addressMode[0] = hipAddressModeClamp;
		texDesc.addressMode[1] = hipAddressModeClamp;
		texDesc.filterMode = hipFilterModePoint;
		texDesc.readMode = hipReadModeElementType;
		texDesc.normalizedCoords = 1;

		hipTextureObject_t* texture = new hipTextureObject_t{};

		hipCreateTextureObject(texture, &resDesc, &texDesc, NULL);

		return texture;
	}

	//-----------------/ buffer init /-----------------//

	void initVertexBuffer(std::vector<Vec3> &buffer) {	
		Vec3* tmp_devicePtr;
		hipMalloc(&tmp_devicePtr, buffer.size() * sizeof(Vec3));
		hipMemcpy(tmp_devicePtr, buffer.data(), buffer.size() * sizeof(Vec3), hipMemcpyHostToDevice);
		hipMemcpyToSymbol(HIP_SYMBOL(CUDA_vertices), &tmp_devicePtr, sizeof(Vec3*));
	}

	void initTexCoordBuffer(std::vector<Vec2> &buffer) {
		Vec3* tmp_devicePtr;
		hipMalloc(&tmp_devicePtr, buffer.size() * sizeof(Vec2));
		hipMemcpy(tmp_devicePtr, buffer.data(), buffer.size() * sizeof(Vec2), hipMemcpyHostToDevice);
		hipMemcpyToSymbol(HIP_SYMBOL(CUDA_texCoords), &tmp_devicePtr, sizeof(Vec2*));
	}

	void initTriangleBuffer(std::vector<Triangle> &buffer) {
		Vec3* tmp_devicePtr;
		hipMalloc(&tmp_devicePtr, buffer.size() * sizeof(Triangle));
		hipMemcpy(tmp_devicePtr, buffer.data(), buffer.size() * sizeof(Triangle), hipMemcpyHostToDevice);
		hipMemcpyToSymbol(HIP_SYMBOL(CUDA_triangles), &tmp_devicePtr, sizeof(Triangle*));
	}

	void initLookupColorBuffer(std::vector<color> &buffer) {
		uint8_t* tmp_devicePtr;
		hipMalloc(&tmp_devicePtr, buffer.size() * sizeof(uint8_t));
		hipMemcpy(tmp_devicePtr, buffer.data(), buffer.size() * sizeof(uint8_t), hipMemcpyHostToDevice);
		hipMemcpyToSymbol(HIP_SYMBOL(CUDA_lookupColors), &tmp_devicePtr, sizeof(uint8_t*));
	}

	void initLookupIndexBuffer(std::vector<uint16_t> &buffer) {
		uint16_t* tmp_devicePtr;
		hipMalloc(&tmp_devicePtr, buffer.size() * sizeof(uint16_t));
		hipMemcpy(tmp_devicePtr, buffer.data(), buffer.size() * sizeof(uint16_t), hipMemcpyHostToDevice);
		hipMemcpyToSymbol(HIP_SYMBOL(CUDA_lookupIndices), &tmp_devicePtr, sizeof(uint16_t*));
	}

	//-----------------/ buffer cleanup /-----------------//

	void freeVertexBuffer() {
		checkCUDA(hipFree(CUDA_vertices));
	}

	void freeTexCoordBuffer() {
		checkCUDA(hipFree(CUDA_texCoords));
	}

	void freeTriangleBuffer() {
		checkCUDA(hipFree(CUDA_triangles));
	}


	void freeLookupColorBuffer() {
		checkCUDA(hipFree(CUDA_lookupColors));
	}

	void freeLookupIndexBuffer() {
		checkCUDA(hipFree(CUDA_lookupIndices));
	}

}

//-----------------/ vector math /-----------------//

__device__ Vec3 add(const Vec3& p1, const Vec3& p2) {
	return {
		p1.x + p2.x,
		p1.y + p2.y,
		p1.z + p2.z
	};
}

__device__ Vec3 sub(const Vec3& p1, const Vec3& p2) {
	return {
		p1.x - p2.x,
		p1.y - p2.y,
		p1.z - p2.z
	};
}

__device__ Vec3 cross(const Vec3& p1, const Vec3& p2) {
	return {
		p1.y * p2.z - p2.y * p1.z,
		p1.z * p2.x - p2.z * p1.x,
		p1.x * p2.y - p2.x * p1.y
	};
}

__device__ float dot(const Vec3& p1, const Vec3& p2) {
	return p1.x * p2.x + p1.y * p2.y + p1.z * p2.z;
}

//-----------------/ AABB triangle collision /-----------------//

__device__ bool cuda_approxTriBoxOverlap(const Vec3& min, const Vec3& max, const Vec3& v0, const Vec3& v1, const Vec3& v2) {
	return !((v0.x > max.x && v1.x > max.x && v2.x > max.x) ||
		(v0.y > max.y && v1.y > max.y && v2.y > max.y) ||
		(v0.z > max.z && v1.z > max.z && v2.z > max.z) ||
		(v0.x < min.x&& v1.x < min.x&& v2.x < min.x) ||
		(v0.y < min.y&& v1.y < min.y&& v2.y < min.y) ||
		(v0.z < min.z&& v1.z < min.z&& v2.z < min.z));
}

__device__ void cuda_findMinMax(float x0, float x1, float x2, float& min, float& max) {
	min = max = x0;
	if (x1 < min)
		min = x1;
	if (x1 > max)
		max = x1;
	if (x2 < min)
		min = x2;
	if (x2 > max)
		max = x2;
}

__device__ bool cuda_planeBoxOverlap(const Vec3& normal, const Vec3& vert, const Vec3& maxbox) {
	Vec3 vmin{ 0, 0, 0 };
	Vec3 vmax{ 0, 0, 0 };

	if (normal.x > 0.0f) {
		vmin.x = -maxbox.x - vert.x;
		vmax.x = maxbox.x - vert.x;
	} else {
		vmin.x = maxbox.x - vert.x;
		vmax.x = -maxbox.x - vert.x;
	}

	if (normal.y > 0.0f) {
		vmin.y = -maxbox.y - vert.y;
		vmax.y = maxbox.y - vert.y;
	} else {
		vmin.y = maxbox.y - vert.y;
		vmax.y = -maxbox.y - vert.y;
	}

	if (normal.z > 0.0f) {
		vmin.z = -maxbox.z - vert.z;
		vmax.z = maxbox.z - vert.z;
	} else {
		vmin.z = maxbox.z - vert.z;
		vmax.z = -maxbox.z - vert.z;
	}

	if (dot(normal, vmin) > 0.0f)
		return false;

	if (dot(normal, vmax) >= 0.0f)
		return true;

	return false;
}

__device__ bool cuda_axisTestX01(float a, float b, float fa, float fb, const Vec3& v0,
	const Vec3& v2, const Vec3& boxhalfsize, float& rad, float& min,
	float& max, float& p0, float& p2) {
	p0 = a * v0.y - b * v0.z;
	p2 = a * v2.y - b * v2.z;
	if (p0 < p2) {
		min = p0;
		max = p2;
	} else {
		min = p2;
		max = p0;
	}
	rad = fa * boxhalfsize.y + fb * boxhalfsize.z;
	if (min > rad || max < -rad)
		return false;
	return true;
}

__device__ bool cuda_axisTestX2(float a, float b, float fa, float fb, const Vec3& v0,
	const Vec3& v1, const Vec3& boxhalfsize, float& rad, float& min,
	float& max, float& p0, float& p1) {
	p0 = a * v0.y - b * v0.z;
	p1 = a * v1.y - b * v1.z;
	if (p0 < p1) {
		min = p0;
		max = p1;
	} else {
		min = p1;
		max = p0;
	}
	rad = fa * boxhalfsize.y + fb * boxhalfsize.z;
	if (min > rad || max < -rad)
		return false;
	return true;
}

__device__ bool cuda_axisTestY02(float a, float b, float fa, float fb, const Vec3& v0,
	const Vec3& v2, const Vec3& boxhalfsize, float& rad, float& min,
	float& max, float& p0, float& p2) {
	p0 = -a * v0.x + b * v0.z;
	p2 = -a * v2.x + b * v2.z;
	if (p0 < p2) {
		min = p0;
		max = p2;
	} else {
		min = p2;
		max = p0;
	}
	rad = fa * boxhalfsize.x + fb * boxhalfsize.z;
	if (min > rad || max < -rad)
		return false;
	return true;
}

__device__ bool cuda_axisTestY1(float a, float b, float fa, float fb, const Vec3& v0,
	const Vec3& v1, const Vec3& boxhalfsize, float& rad, float& min,
	float& max, float& p0, float& p1) {
	p0 = -a * v0.x + b * v0.z;
	p1 = -a * v1.x + b * v1.z;
	if (p0 < p1) {
		min = p0;
		max = p1;
	} else {
		min = p1;
		max = p0;
	}
	rad = fa * boxhalfsize.x + fb * boxhalfsize.z;
	if (min > rad || max < -rad)
		return false;
	return true;
}

__device__ bool cuda_axisTestZ12(float a, float b, float fa, float fb, const Vec3& v1,
	const Vec3& v2, const Vec3& boxhalfsize, float& rad, float& min,
	float& max, float& p1, float& p2) {
	p1 = a * v1.x - b * v1.y;
	p2 = a * v2.x - b * v2.y;
	if (p1 < p2) {
		min = p1;
		max = p2;
	} else {
		min = p2;
		max = p1;
	}
	rad = fa * boxhalfsize.x + fb * boxhalfsize.y;
	if (min > rad || max < -rad)
		return false;
	return true;
}

__device__ bool cuda_axisTestZ0(float a, float b, float fa, float fb, const Vec3& v0,
	const Vec3& v1, const Vec3& boxhalfsize, float& rad, float& min,
	float& max, float& p0, float& p1) {
	p0 = a * v0.x - b * v0.y;
	p1 = a * v1.x - b * v1.y;
	if (p0 < p1) {
		min = p0;
		max = p1;
	} else {
		min = p1;
		max = p0;
	}
	rad = fa * boxhalfsize.x + fb * boxhalfsize.y;
	if (min > rad || max < -rad)
		return false;
	return true;
}

__device__ bool cuda_triBoxOverlap(const Vec3& boxcenter, const Vec3& boxhalfsize, const Vec3& tv0, const Vec3& tv1, const Vec3& tv2) {

	Vec3 v0, v1, v2;
	float min, max, p0, p1, p2, rad, fex, fey, fez;
	Vec3 normal, e0, e1, e2;

	v0 = sub(tv0, boxcenter);
	v1 = sub(tv1, boxcenter);
	v2 = sub(tv2, boxcenter);

	e0 = sub(v1, v0);
	e1 = sub(v2, v1);
	e2 = sub(v0, v2);

	fex = fabsf(e0.x);
	fey = fabsf(e0.y);
	fez = fabsf(e0.z);

	if (!cuda_axisTestX01(e0.z, e0.y, fez, fey, v0, v2, boxhalfsize, rad, min, max, p0, p2))
		return false;
	if (!cuda_axisTestY02(e0.z, e0.x, fez, fex, v0, v2, boxhalfsize, rad, min, max, p0, p2))
		return false;
	if (!cuda_axisTestZ12(e0.y, e0.x, fey, fex, v1, v2, boxhalfsize, rad, min, max, p1, p2))
		return false;

	fex = fabsf(e1.x);
	fey = fabsf(e1.y);
	fez = fabsf(e1.z);

	if (!cuda_axisTestX01(e1.z, e1.y, fez, fey, v0, v2, boxhalfsize, rad, min, max, p0, p2))
		return false;
	if (!cuda_axisTestY02(e1.z, e1.x, fez, fex, v0, v2, boxhalfsize, rad, min, max, p0, p2))
		return false;
	if (!cuda_axisTestZ0(e1.y, e1.x, fey, fex, v0, v1, boxhalfsize, rad, min, max, p0, p1))
		return false;

	fex = fabsf(e2.x);
	fey = fabsf(e2.y);
	fez = fabsf(e2.z);

	if (!cuda_axisTestX2(e2.z, e2.y, fez, fey, v0, v1, boxhalfsize, rad, min, max, p0, p1))
		return false;
	if (!cuda_axisTestY1(e2.z, e2.x, fez, fex, v0, v1, boxhalfsize, rad, min, max, p0, p1))
		return false;
	if (!cuda_axisTestZ12(e2.y, e2.x, fey, fex, v1, v2, boxhalfsize, rad, min, max, p1, p2))
		return false;

	cuda_findMinMax(v0.x, v1.x, v2.x, min, max);
	if (min > boxhalfsize.x || max < -boxhalfsize.x)
		return false;

	cuda_findMinMax(v0.y, v1.y, v2.y, min, max);
	if (min > boxhalfsize.y || max < -boxhalfsize.y)
		return false;

	cuda_findMinMax(v0.z, v1.z, v2.z, min, max);
	if (min > boxhalfsize.z || max < -boxhalfsize.z)
		return false;

	normal = cross(e0, e1);
	if (!cuda_planeBoxOverlap(normal, v0, boxhalfsize))
		return false;

	return true;
}

//-----------------/ main kernel /-----------------//

__global__ void chunkInserter(const size_t* indexBuffer, const size_t numIndices, hipTextureObject_t tex, uint16_t* blockBuffer, const int chunkX, const int chunkZ) {

	const size_t index = (size_t)threadIdx.x + (size_t)blockIdx.x * (size_t)blockDim.x;

	const Vec3 boxCenter = {
		chunkX * 16.0f + index % 16 + 0.5f,
		(index / 4096 + CUDA_minChunkY) * 16.0f + (index % 4096) / 256 + 0.5f,
		chunkZ * 16.0f + (index / 16) % 16 + 0.5f
	};

	const Vec3 boxHalfSize = { 0.5f, 0.5f, 0.5f };
	const Vec3 boxMin = sub(boxCenter, boxHalfSize);
	const Vec3 boxMax = add(boxCenter, boxHalfSize);

	for (size_t i = 0; i < numIndices; i++) {
		const Triangle* tri = &CUDA_triangles[indexBuffer[i]];

		if (cuda_approxTriBoxOverlap(boxMin, boxMax, CUDA_vertices[tri->vertexIndices[0]], CUDA_vertices[tri->vertexIndices[1]], CUDA_vertices[tri->vertexIndices[2]]) &&
			cuda_triBoxOverlap(boxCenter, boxHalfSize, CUDA_vertices[tri->vertexIndices[0]], CUDA_vertices[tri->vertexIndices[1]], CUDA_vertices[tri->vertexIndices[2]])){

			if (tri->blockID == UINT16_MAX) {
				const Vec3 s = sub(CUDA_vertices[tri->vertexIndices[1]], CUDA_vertices[tri->vertexIndices[0]]);

				const Vec3 t = sub(CUDA_vertices[tri->vertexIndices[2]], CUDA_vertices[tri->vertexIndices[0]]);

				const Vec3 n = cross(s, t);

				const Vec3 delta = sub(boxCenter, CUDA_vertices[tri->vertexIndices[0]]);

				const float nnInv = 1.0f / dot(n, n);
				const float w = dot(cross(s, delta), n) * nnInv;
				const float v = dot(cross(delta, t), n) * nnInv;
				const float u = 1.0f - w - v;

				const float x = u * CUDA_texCoords[tri->vertexIndices[0]].u + v * CUDA_texCoords[tri->vertexIndices[1]].u + w * CUDA_texCoords[tri->vertexIndices[2]].u;
				const float y = u * CUDA_texCoords[tri->vertexIndices[0]].v + v * CUDA_texCoords[tri->vertexIndices[1]].v + w * CUDA_texCoords[tri->vertexIndices[2]].v;

				const uchar4 color = tex2D<uchar4>(tex, x, y);

				size_t closestMatch = 0;
				uint64_t minDelta = UINT64_MAX;
				for (size_t j = 0; j < CUDA_lookupSize; j++) {

					const uint64_t delta = (uint64_t)abs(color.x - CUDA_lookupColors[j].r) +
						(uint64_t)abs(color.y - CUDA_lookupColors[j].g) +
						(uint64_t)abs(color.z - CUDA_lookupColors[j].b) +
						(uint64_t)abs(color.w - CUDA_lookupColors[j].a);

					if (delta < minDelta) {
						minDelta = delta;
						closestMatch = j;
					}
				}

				blockBuffer[index] = CUDA_lookupIndices[closestMatch];
			} else {
				blockBuffer[index] = tri->blockID;
			}
			break;
		}
	}
}

namespace CUDA {
	void insertBlocks(size_t numBlocks, size_t numThreads, size_t* indexBuffer, size_t numIndices, hipTextureObject_t tex, uint16_t* blockBuffer, int chunkX, int chunkZ) {
		chunkInserter <<<numBlocks, numThreads>>> (indexBuffer, numIndices, tex, blockBuffer, chunkX, chunkZ);
	}
}
